
#include <hip/hip_runtime.h>
#include <iostream>
using std::cout;
using std::endl;

__global__ void matAdd(int *a, int *b, int *c)
{
    int index = blockIdx.x * gridDim.y + blockIdx.y;
    c[index] = a[index] + b[index];
}

void print_matrix(int *matrix, int M, int N)
{
    for(int i = 0; i < M; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            cout << matrix[i * N + j] << " ";
        }
        cout << endl;
    }
}

int main()
{
    int M = 7; // num rows
    int N = 3; // num cols

    int size = M * N * sizeof(int);

    // Allocate memory for new matrices
    int *a = new int[size];
    int *b = new int[size];
    int *c = new int[size];

    for(int i = 0; i < M; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            a[i * N + j] = i;
            b[i * N + j] = j;
        }
    }

    int *d_a;
    int *d_b;
    int *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 gridSize(M, N);
    matAdd<<<gridSize, 1>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    print_matrix(a, M, N);
    cout << endl;
    print_matrix(b, M, N);
    cout << endl;
    print_matrix(c, M, N);
    cout << endl;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}