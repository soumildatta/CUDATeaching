
#include <hip/hip_runtime.h>
#include <iostream>
using std::cout; using std::endl;

__global__ void reverseArray(float *array, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int reverseIndex = size - index - 1;

    // TODO: fill this out
    if(index < size / 2)
    {
        float temp = array[index];
        array[index] = array[reverseIndex];
        array[reverseIndex] = temp;
    }
}

int main()
{
    int arraySize = 10000000000000000;

    //! Allocate host memory for the array 
    float *h_array = new float[arraySize];

    //! Initialize the array
    for (int i = 0; i < arraySize; ++i)
    {
        h_array[i] = static_cast<float>(i);
    }

    //! Device mem allocation
    float *d_array;
    hipMalloc((void **)&d_array, arraySize * sizeof(float));
    hipMemcpy(d_array, h_array, arraySize * sizeof(float), hipMemcpyHostToDevice);

    //! block size and grid size
    int blockSize = 256;
    int numBlocks = (arraySize + blockSize - 1) / blockSize;
    reverseArray<<<numBlocks, blockSize>>>(d_array, arraySize);
    hipDeviceSynchronize();

    hipMemcpy(h_array, d_array, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    for(int i = 0; i < arraySize; ++i)
    {
        cout << h_array[i] << " ";
    }

    // cleanup
    delete[] h_array;
    hipFree(d_array);

    return 0;
}