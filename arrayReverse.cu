#include <iostream>
#include <hip/hip_runtime.h>

// Kernel function to reverse an array on the GPU
__global__ void reverseArray(float* array, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int reverseIndex = size - index - 1;

    if (index < size / 2) {
        float temp = array[index];
        array[index] = array[reverseIndex];
        array[reverseIndex] = temp;
    }
}

int main() {
    // Size of the array
    int arraySize = 10;

    // Allocate memory for the array on the host
    float* hostArray = new float[arraySize];

    // Initialize the array
    for (int i = 0; i < arraySize; ++i) {
        hostArray[i] = static_cast<float>(i);
    }

    // Allocate memory for the array on the device
    float* deviceArray;
    hipMalloc((void**)&deviceArray, arraySize * sizeof(float));

    // Copy the array from host to device
    hipMemcpy(deviceArray, hostArray, arraySize * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    int blockSize = 256;
    int gridSize = (arraySize + blockSize - 1) / blockSize;
    reverseArray<<<gridSize, blockSize>>>(deviceArray, arraySize);

    // Copy the reversed array back to the host
    hipMemcpy(hostArray, deviceArray, arraySize * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    delete[] hostArray;
    hipFree(deviceArray);

    // Print the reversed array
    std::cout << "Reversed array: ";
    for (int i = 0; i < arraySize; ++i) {
        std::cout << hostArray[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}
