
#include <hip/hip_runtime.h>
#include<iostream>
using std::cout;
using std::endl;

__global__ void printKernel()
{
    printf("%d", threadIdx.x);
}

int main()
{
    // cout << "Hello World" << endl;
    printKernel<<<1, 10>>>();
    hipDeviceSynchronize();

    return 0;
}