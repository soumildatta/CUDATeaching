
#include <hip/hip_runtime.h>
#include <iostream>
using std::cout;
using std::endl;

__global__ void incrementVariable(int* result) {
    atomicAdd(result, 1);
}

int main() {

    int hostResult = 0;
    int* deviceResult;
    hipMalloc((void**)&deviceResult, sizeof(int));
    hipMemcpy(deviceResult, &hostResult, sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    int numThreads = 256;
    incrementVariable<<<1, numThreads>>>(deviceResult);

    // Copy the result back to the host
    hipMemcpy(&hostResult, deviceResult, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceResult);

    // Print the result
    cout << "Final value: " << hostResult << endl;

    return 0;
}