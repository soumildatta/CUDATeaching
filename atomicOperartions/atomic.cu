#include <iostream>
using std::cout;
using std::endl;

int main()
{
    size_t size = 500000000;

    //! HOST DATA INITIALIZATION
    // Allocate array on CPU
    float *h_array = (float *)malloc(size);
    // Add values to array on CPU
    for(int i = 0; i < size; ++i)
    {
        h_array[i] = (float)(rand()) / (float)(rand());
    }

    //! DEVICE INITIALIZATION
    float *d_array = NULL;
    hipMalloc((void **)&d_array, size);
    hipMemcpy(d_array, h_array, size, hipMemcpyHostToDevice);

    float *result = (float *)malloc(sizeof(float));
    result[0] = 0;
    
    float *d_result = NULL;
    hipMalloc((void **)&result, sizeof(float));
    hipMemcpy(result, sizeof(float), hipMemcpyHostToDevice);

    return 0;
}