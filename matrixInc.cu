#include "hip/hip_runtime.h"
/*
Program to add 2 matrics of size M * N in CUDA C++
Using 2-D grid of M*N size (i.e. grid contains M*N blocks arranged in 2D fashion)
Each block contains 1 thread
*/
#include<iostream>
using std::cout; using std::endl;

#define M 7
#define N 3

__global__ void matAdd(int* a, const int increment)
{
    int idx = blockIdx.x * gridDim.y + blockIdx.y;
    a[idx] = a[idx] + increment;
}

__host__ void print_matrix(int* matrix)
{
    for(int i=0; i<M; ++i)
    {
        for(int j=0; j<N; ++j)
        {
            cout<<matrix[i*N+j]<<' ';
        }
        cout<<"\n";
    }
    cout<<"\n";
}

int main()
{
    int size = M * N * sizeof(int);
    int* a = new int[size];

    for(int i=0; i<M; ++i)
    {
        for(int j=0; j<N; ++j)
        {
            a[i*N + j] = i; //Fill your own values here
        }
    }

    print_matrix(a);

    /* Setting up variables on device. i.e. GPU */
    int *d_a;
    hipMalloc((void**)&d_a, size);

    /* Copy data from host to device */
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    dim3 gridSize(M, N);
    matAdd<<<gridSize, 1>>>(d_a, 3);
    hipDeviceSynchronize();

    /* Copy result from GPU device to host */
    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
    
    /* Print result */
    print_matrix(a);

    /* Cleanup device and host memory */
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete a;
    delete b;
    delete c;

    return 0;
}