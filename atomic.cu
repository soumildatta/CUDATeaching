
#include <hip/hip_runtime.h>
#include <iostream>
using std::cout;
using std::endl;

__global__ void add(int *result)
{
    // result += 1;
    atomicAdd(result, 1);
}

int main()
{
    int h_result = 0;

    int *d_result;
    hipMalloc((void **)&d_result, sizeof(int));
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    int numThreads = 256;
    int numBlocks = 1;

    //! Kernel Invocation
    add<<<numBlocks, numThreads>>>(d_result);

    //! Device Sync
    hipDeviceSynchronize();

    //! Memcopy back to host 
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    //! Free memory
    hipFree(d_result);

    //! Print result
    cout << "Final result: " << h_result << endl;

    return 0;
}